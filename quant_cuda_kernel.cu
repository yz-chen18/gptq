#include "hip/hip_runtime.h"
#include <torch/all.h>
#include <torch/python.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

template <typename scalar_t>
__global__ void VecQuant8MatMulKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  scalar_t* __restrict__ zeros,
    int height,
    int width
);

__global__ void VecQuant8MatMulKernelFaster(
    const  char2* __restrict__ vec,
    const    int* __restrict__ mat,
           int* __restrict__ mul,
    int height,
    int width
);

__global__ void BatchVecQuant8MatMulKernelFaster(
    const  char2* __restrict__ vec,
    const    int* __restrict__ mat,
           int* __restrict__ mul,
    int height,
    int width,
    int batchsize
);

const int BLOCKWIDTH  = 256;
const int BLOCKHEIGHT =  64;

void vecquant8matmul_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros
) {
  int height = mat.size(0);
  int width = mat.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT - 1) / BLOCKHEIGHT,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH
  );
  dim3 threads(BLOCKWIDTH);

  AT_DISPATCH_FLOATING_TYPES(
    vec.type(), "vecquant8matmul_cuda", ([&] {
      VecQuant8MatMulKernel<<<blocks, threads>>>(
        vec.data<scalar_t>(), mat.data<int>(), mul.data<scalar_t>(),
        scales.data<scalar_t>(), zeros.data<scalar_t>(),
        height, width
      );
    })
  );
}

void vecquant8matmul_faster_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros
) {
  int batchsize = (vec.dim() == 1) ? 1 : vec.size(0);
  int height = mat.size(0);
  int width = mat.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT - 1) / BLOCKHEIGHT,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH
  );
  dim3 threads(BLOCKWIDTH);

  if (batchsize == 1) {
    VecQuant8MatMulKernelFaster<<<blocks, threads>>>(
      (char2*) vec.data_ptr(),
      mat.data_ptr<int>(),
      mul.data_ptr<int>(),
      height, width
    );
  } else {
    BatchVecQuant8MatMulKernelFaster<<<blocks, threads>>>(
      (char2*) vec.data_ptr(),
      mat.data_ptr<int>(),
      mul.data_ptr<int>(),
      height, width, batchsize
    );
  }
  
}

__device__ inline unsigned int as_unsigned(int i) {
  return *reinterpret_cast<unsigned int*>(&i);
}

template <typename scalar_t>
__global__ void VecQuant8MatMulKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  scalar_t* __restrict__ zeros,
    int height,
    int width
) {
  int row = BLOCKHEIGHT * blockIdx.x;
  int col =  BLOCKWIDTH * blockIdx.y + threadIdx.x;

  __shared__ scalar_t blockvec[BLOCKWIDTH];
  blockvec[threadIdx.x] = vec[(row / BLOCKHEIGHT) * BLOCKWIDTH + threadIdx.x];
  __syncthreads();

  scalar_t scale = scales[col];
  scalar_t zero = zeros[col];

  scalar_t res = 0;
  int i = width * row + col;
  int k = 0;

  unsigned int tmp1;
  unsigned int tmp2;
  unsigned int tmp;

  while (k < BLOCKWIDTH) {
    tmp1 = as_unsigned(mat[i]);
    res += (scale * scalar_t((tmp1 >>  0) & 0x7) - zero) * blockvec[k + 0];
    res += (scale * scalar_t((tmp1 >>  3) & 0x7) - zero) * blockvec[k + 1];
    res += (scale * scalar_t((tmp1 >>  6) & 0x7) - zero) * blockvec[k + 2];
    res += (scale * scalar_t((tmp1 >>  9) & 0x7) - zero) * blockvec[k + 3];
    res += (scale * scalar_t((tmp1 >> 12) & 0x7) - zero) * blockvec[k + 4];
    res += (scale * scalar_t((tmp1 >> 15) & 0x7) - zero) * blockvec[k + 5];
    res += (scale * scalar_t((tmp1 >> 18) & 0x7) - zero) * blockvec[k + 6];
    res += (scale * scalar_t((tmp1 >> 21) & 0x7) - zero) * blockvec[k + 7];
    res += (scale * scalar_t((tmp1 >> 24) & 0x7) - zero) * blockvec[k + 8];
    res += (scale * scalar_t((tmp1 >> 27) & 0x7) - zero) * blockvec[k + 9];
    i += width;
    tmp2 = as_unsigned(mat[i]);
    tmp = (tmp1 >> 30) | ((tmp2 << 2) & 0x4);
    tmp2 >>= 1;
    res += (scale * scalar_t(tmp) - zero) * blockvec[k + 10];
    k += 11;
    res += (scale * scalar_t((tmp2 >>  0) & 0x7) - zero) * blockvec[k + 0];
    res += (scale * scalar_t((tmp2 >>  3) & 0x7) - zero) * blockvec[k + 1];
    res += (scale * scalar_t((tmp2 >>  6) & 0x7) - zero) * blockvec[k + 2];
    res += (scale * scalar_t((tmp2 >>  9) & 0x7) - zero) * blockvec[k + 3];
    res += (scale * scalar_t((tmp2 >> 12) & 0x7) - zero) * blockvec[k + 4];
    res += (scale * scalar_t((tmp2 >> 15) & 0x7) - zero) * blockvec[k + 5];
    res += (scale * scalar_t((tmp2 >> 18) & 0x7) - zero) * blockvec[k + 6];
    res += (scale * scalar_t((tmp2 >> 21) & 0x7) - zero) * blockvec[k + 7];
    res += (scale * scalar_t((tmp2 >> 24) & 0x7) - zero) * blockvec[k + 8];
    res += (scale * scalar_t((tmp2 >> 27) & 0x7) - zero) * blockvec[k + 9];
    i += width;
    tmp1 = as_unsigned(mat[i]);
    tmp = (tmp2 >> 30) | ((tmp1 << 1) & 0x6);
    tmp1 >>= 2;
    res += (scale * scalar_t(tmp) - zero) * blockvec[k + 10];
    k += 11;
    res += (scale * scalar_t((tmp1 >>  0) & 0x7) - zero) * blockvec[k + 0];
    res += (scale * scalar_t((tmp1 >>  3) & 0x7) - zero) * blockvec[k + 1];
    res += (scale * scalar_t((tmp1 >>  6) & 0x7) - zero) * blockvec[k + 2];
    res += (scale * scalar_t((tmp1 >>  9) & 0x7) - zero) * blockvec[k + 3];
    res += (scale * scalar_t((tmp1 >> 12) & 0x7) - zero) * blockvec[k + 4];
    res += (scale * scalar_t((tmp1 >> 15) & 0x7) - zero) * blockvec[k + 5];
    res += (scale * scalar_t((tmp1 >> 18) & 0x7) - zero) * blockvec[k + 6];
    res += (scale * scalar_t((tmp1 >> 21) & 0x7) - zero) * blockvec[k + 7];
    res += (scale * scalar_t((tmp1 >> 24) & 0x7) - zero) * blockvec[k + 8];
    res += (scale * scalar_t((tmp1 >> 27) & 0x7) - zero) * blockvec[k + 9];
    i += width;
    k += 10;
  }

  atomicAdd(&mul[col], res);
}

__global__ void VecQuant8MatMulKernelFaster(
    const  char2* __restrict__ vec,
    const    int* __restrict__ mat,
           int* __restrict__ mul,
    int height,
    int width
) {
  const int blockwidth2 = BLOCKWIDTH / 2;

  int row = BLOCKHEIGHT * blockIdx.x;
  int col = BLOCKWIDTH * blockIdx.y + threadIdx.x;

  __shared__ char2 blockvec[blockwidth2];
  if (threadIdx.x < blockwidth2)
    blockvec[threadIdx.x] = vec[(row / BLOCKHEIGHT) * blockwidth2 + threadIdx.x];
  
  int i = width * row + col;
  int k = 0;

  int res = 0;

  __syncthreads();

  unsigned int tmp;

  while (k < blockwidth2) {
    tmp = as_unsigned(mat[i]);
    res += ((tmp >> 0) & 0xff) * blockvec[k + 0].x;
    res += ((tmp >> 8) & 0xff) * blockvec[k + 0].y;
    res += ((tmp >> 16) & 0xff) * blockvec[k + 1].x;
    res += ((tmp >> 24) & 0xff) * blockvec[k + 1].y;
    i += width;
    k += 2;
  }

  atomicAdd(&mul[col], res);
}

__global__ void BatchVecQuant8MatMulKernelFaster(
    const  char2* __restrict__ vec,
    const    int* __restrict__ mat,
           int* __restrict__ mul,
    int height,
    int width,
    int batchsize
) {
  const int blockwidth2 = BLOCKWIDTH / 2;

  int row = BLOCKHEIGHT * blockIdx.x;
  int col = BLOCKWIDTH * blockIdx.y + threadIdx.x;

  __shared__ char2 blockvec[blockwidth2];

  for (int b = 0; b < batchsize; b++) {
    if (threadIdx.x < blockwidth2)
      blockvec[threadIdx.x] = vec[(row / BLOCKHEIGHT) * blockwidth2 + threadIdx.x + b * height / 8 * 16];
    int i = width * row + col;
    int k = 0;

    int res = 0;

    __syncthreads();

    unsigned int tmp;

    while (k < blockwidth2) {
      tmp = as_unsigned(mat[i]);
      res += ((tmp >> 0) & 0xff) * blockvec[k + 0].x;
      res += ((tmp >> 8) & 0xff) * blockvec[k + 0].y;
      res += ((tmp >> 16) & 0xff) * blockvec[k + 1].x;
      res += ((tmp >> 24) & 0xff) * blockvec[k + 1].y;
      i += width;
      k += 2;
    }

    atomicAdd(&mul[col + b * width], res);
  }
}